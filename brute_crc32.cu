#include <time.h>
#include "stdio.h"
#include "hip/hip_runtime.h"

#define BlockNum 256
#define ThreadNum 1024 
#define Len 4

__host__ __device__ unsigned int crc32(unsigned char *message)
{
   int i, j;
   unsigned int byte, crc, mask;
   i = 0;
   crc = 0xFFFFFFFF;
   while (message[i] != 0)
   {
      byte = message[i];       // Get next byte.
      crc = crc ^ byte;
      for (j = 7; j >= 0; j--) // Do eight times.
      {    
         mask = -(crc & 1);
         crc = (crc >> 1) ^ (0xEDB88320 & mask);
      }
      i = i + 1;
   }
   return ~crc;
}

__host__ void crc32Host(int len, unsigned int target)
{
	unsigned char buf[Len];
	for(int i=0;i<len;i++)
	{
		buf[i]=0;
	}
	unsigned int crc=0;
	while(target!=crc)
	{
		buf[0]++;
       	for(int i=0;i<len;i++)
       	{
       		if (buf[i]>=255)
       		{
       			buf[(i+1)%len]++;
       			buf[i]=0;
       		}
   		}
        crc=crc32(buf);
    	if(crc == target)
    	{
    		printf("Input Found in CPU=");
    		for (int i = 0; i < Len; ++i)
    		{
    			printf("%c",buf[i]);
    		}
    		printf("\n");
    		break;
		}
	}
}

__global__ void crc32Device(int len, unsigned int target)
{ 
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int size = BlockNum*ThreadNum;
	unsigned long long spacesearch=1;
	for(int i=0;i<len;i++)
	{
		spacesearch *=256;
	}
	if(idx==0) printf("spacesearc=%ld,Size=%d\n",spacesearch,size );
	{
		__syncthreads();
	}
	unsigned char buf[Len];
	for(int i=0;i<len;i++)
	{
		buf[i]=0;
	}
	unsigned int crc=0;
	unsigned int index=idx*((spacesearch/size)+1);
	for(int i=0;i<Len;i++)
	{
		buf[i]=(unsigned char)((index)&0xff);
	   	index=(index) >>8;
	}
	for(int i=0;i<((spacesearch/size)+1);i++)
	{
		for(int j=0;j<len;j++)
		{
			if (buf[j]>=255)
			{
				buf[(j+1)%len]++;
	       		buf[j]=0;
	       	}	
	   	}
        crc=crc32(buf);
        buf[0]++;
   		if(crc == target)
   		{
   			printf("Input Found in GPU=");
    		for (int i = 0; i < Len; ++i)
    		{
    			printf("%c",buf[i]);
 			}
    		printf("\n");
		}
    }
	__syncthreads(); 
}

int main()
{
	unsigned char boi[Len]={0};
	for(int i=0;i<Len;i++)
		boi[i]='b';
	unsigned int test =crc32(boi);
	printf("%x\n",test );

    // Set the Device Number
    hipSetDevice(0);

    // Allocating memory in device
    int len; unsigned int target;
    hipMalloc((void**)&len, sizeof(int) * 1);
    hipMalloc((void**)&target, sizeof(unsigned int) * 1);

    // Setting CUDA timer finction
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host function in CPU
    hipEventRecord(start,0);

	// crc32Host( Len,test);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float miliseconds_cpu = 0;
    hipEventElapsedTime(&miliseconds_cpu,start,stop);

  	// printf("Elapsed Time for the CPU computation is :%f\n",miliseconds_cpu/1000);
    // Device function in GPU
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float miliseconds_gpu = 0;
   	hipEventRecord(start,0);
    crc32Device<<<BlockNum,ThreadNum>>>(Len, test);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&miliseconds_gpu,start,stop);
    printf("Elapsed Time for the GPU computation is :%f\n",miliseconds_gpu/1000);
	
	//printf("GPU speedup over CPU is :%f\nx",miliseconds_cpu/miliseconds_gpu);
    hipDeviceReset();
    return 0;
}